#include "hip/hip_runtime.h"
#include "hConstTypes.h"
#include "hgeneralCPU.h"
#include "hgeneralGPU.h"
#include "hIncidentWaveGPU.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

// Incident wave in Fourier Space
__global__ void IncidentWaveFS(sGP GP, sLens Lens, double x, double y, double2 *Psig){
	int j = threadIdx.x + blockIdx.x*blockDim.x;
	int i = threadIdx.y + blockIdx.y*blockDim.y;

	if ((i < GP.nx)&&(j < GP.ny)){	
		int k = i*GP.ny+j;
		double gx = ((i<GP.nxh)?i:(i-GP.nx))*GP.dgx;
		double gy = ((j<GP.nyh)?j:(j-GP.ny))*GP.dgy;
		double g2 = gx*gx + gy*gy;
		if ((Lens.gmin2 <= g2)&&(g2 < Lens.gmax2)){
			double chi = x*gx + y*gy + g2*(Lens.cCs5*g2*g2+Lens.cCs3*g2+Lens.cf);
			if ((Lens.m!=0)||(Lens.cmfa2!=0)||(Lens.cmfa3!=0)){
				double g = sqrt(g2);
				double phi = atan(gy/gx);
				chi += Lens.m*phi + Lens.cmfa2*g2*sin(2*(phi-Lens.afa2)) + Lens.cmfa3*g*g2*sin(3*(phi-Lens.afa3));				
			}
			sincos(chi, &gy , &gx);		
			Psig[k].x = gx; 
			Psig[k].y = gy;	
		}
		else{
 			Psig[k].x = 0.0; 
			Psig[k].y = 0.0; 
		}
	}
}

void cIncidentWaveGPU::freeMemory(){
	hipDeviceSynchronize(); // wait to finish the work in the GPU

	fsGP_Init(GP);
	fsBT_Init(BT);
	fsLens_Init(Lens);

	cudaFreen(Sd);
	PlanPsi = 0;
}

cIncidentWaveGPU::cIncidentWaveGPU(){
	fsGP_Init(GP);
	fsBT_Init(BT);
	fsLens_Init(Lens);

	Sd = 0;
	PlanPsi = 0;
}

cIncidentWaveGPU::~cIncidentWaveGPU(){
	freeMemory();
}

void cIncidentWaveGPU::SetInputData(sBT &BT_i, sGP &GP_i, sLens &Lens_i, hipfftHandle &PlanPsi_i){
	freeMemory();

	BT = BT_i;
	GP = GP_i;
	Lens = Lens_i;
	PlanPsi = PlanPsi_i;
	hipMalloc((void**)&Sd, 2*thrnxy*cSizeofRD);
}

void cIncidentWaveGPU::Psi0(double2 *&Psig){
	SetValueVectorC(BT.Bnxy, BT.Tnxy, GP.nxy, 1.0, 0.0, Psig);
}

void cIncidentWaveGPU::Psi0(double x, double y, double2 *&Psig){
	IncidentWaveFS<<<BT.Bnxny, BT.Tnxny>>>(GP, Lens, c2Pi*x, c2Pi*y, Psig);
	hipfftExecZ2Z(PlanPsi, Psig, Psig, HIPFFT_BACKWARD);
	double Totalsum = SumAc2(BT.Bnxy, BT.Tnxy, GP.nxy, Psig, Sd);
	ScaleVectorC(BT.Bnxy, BT.Tnxy, GP.nxy, sqrt(GP.nxy/Totalsum), Psig);
}