#include "math.h"
#include <memory.h>
#include "..\General\hConstTypes.h"
#include "..\General\hgeneralCPU.h"
#include "..\General\hMatlab2Cpp.h"
#include "..\General\hMulSliGPU.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include "hipfft/hipfft.h"
#include "mex.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
	sInMSTEM InMSTEM;
	sComplex aPsih ;
	double *M2aPsih, *aM2Psih;

	f_InMulSli_Init(InMSTEM);
	Matlab2InMulSli(prhs[0], InMSTEM);

	cMulSliGPU MulSliGPU;
	MulSliGPU.SetInputData(InMSTEM);

	switch (InMSTEM.SimType){
		case 1:
			MulSliGPU.Cal_STEM();
			ImSTEM2Matlab(MulSliGPU.STEM.nThk, MulSliGPU.STEM.nDet, MulSliGPU.STEM.line, MulSliGPU.STEM.nxs, MulSliGPU.STEM.nys, MulSliGPU.STEM.ImSTEM, plhs[0]);
			break;
		case 2:		
			plhs[0] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxCOMPLEX);
			aPsih.real = mxGetPr(plhs[0]); aPsih.imag = mxGetPi(plhs[0]);
			plhs[1] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxREAL);
			aM2Psih = mxGetPr(plhs[1]);
			MulSliGPU.Cal_CBED(aPsih, aM2Psih);
			break;
		case 3:						
			plhs[0] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxCOMPLEX);
			aPsih.real = mxGetPr(plhs[0]); aPsih.imag = mxGetPi(plhs[0]);
			plhs[1] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxREAL);
			M2aPsih = mxGetPr(plhs[1]);
			plhs[2] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxREAL);
			aM2Psih = mxGetPr(plhs[2]);
			MulSliGPU.Cal_HRTEM(aPsih, M2aPsih, aM2Psih);
			break;
		case 4:						
			plhs[0] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxCOMPLEX);
			aPsih.real = mxGetPr(plhs[0]); aPsih.imag = mxGetPi(plhs[0]);
			plhs[1] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxREAL);
			aM2Psih = mxGetPr(plhs[1]);
			MulSliGPU.Cal_ED(aPsih, aM2Psih);
			break;
		case 5:						

			break;
		case 6:						

			break;
		case 10:						
			plhs[0] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxCOMPLEX);
			aPsih.real = mxGetPr(plhs[0]); aPsih.imag = mxGetPi(plhs[0]);
			plhs[1] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxREAL);
			aM2Psih = mxGetPr(plhs[1]);
			MulSliGPU.Cal_ExitWaveRS(aPsih, aM2Psih);
			break;
		case 11:						
			plhs[0] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxCOMPLEX);
			aPsih.real = mxGetPr(plhs[0]); aPsih.imag = mxGetPi(plhs[0]);
			plhs[1] = mxCreateDoubleMatrix(InMSTEM.ny, InMSTEM.nx, mxREAL);
			aM2Psih = mxGetPr(plhs[1]);
			MulSliGPU.Cal_ExitWaveFS(aPsih, aM2Psih);
			break;
	}

	f_InMulSli_Free(InMSTEM);
}