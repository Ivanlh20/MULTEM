#include "hip/hip_runtime.h"
/*
 * This file is part of MULTEM.
 * Copyright 2020 Ivan Lobato <Ivanlh20@gmail.com>
 * Copyright 2021 Diamond Light Source
 * Copyright 2021 Rosalind Franklin Institute
 *
 * Author: James Parkhurst
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http:// www.gnu.org/licenses/>.
 */

#include <iostream>
#include <multem.h>
#include <multem/error.h>
#include <types.cuh>
#include <input_multislice.cuh>
#include <output_multislice.hpp>
#include <tem_simulation.cuh>

namespace mt {
  
  /****************************************************************************
   * The SystemConfiguration interface
   ***************************************************************************/

  struct SystemConfiguration::Data {
    System_Configuration data;
    Data() {}
    Data(const System_Configuration& d)
        : data(d) {}
  };

  SystemConfiguration::SystemConfiguration()
      : impl_(std::make_unique<Data>()) {}

  SystemConfiguration::SystemConfiguration(const SystemConfiguration& other)
      : impl_(std::make_unique<Data>(*other.impl_)) {}

  SystemConfiguration::SystemConfiguration(SystemConfiguration&& other) = default;

  SystemConfiguration& SystemConfiguration::operator=(const SystemConfiguration& other) {
    *impl_ = *other.impl_;
    return *this;
  }

  SystemConfiguration::SystemConfiguration(const Data& other)
      : impl_(std::make_unique<Data>(other)) {}

  SystemConfiguration& SystemConfiguration::operator=(SystemConfiguration&&) = default;

  SystemConfiguration::~SystemConfiguration() = default;

  const SystemConfiguration::Data& SystemConfiguration::internal() const {
    return *impl_;
  }

  void SystemConfiguration::set_precision(ePrecision precision) {
    impl_->data.precision = precision;
  }

  ePrecision SystemConfiguration::get_precision() const {
    return impl_->data.precision;
  }

  void SystemConfiguration::set_device(eDevice device) {
    impl_->data.device = device;
  }

  eDevice SystemConfiguration::get_device() const {
    return impl_->data.device;
  }

  void SystemConfiguration::set_cpu_ncores(int cpu_ncores) {
    impl_->data.cpu_ncores = cpu_ncores;
  }

  int SystemConfiguration::get_cpu_ncores() const {
    return impl_->data.cpu_ncores;
  }

  void SystemConfiguration::set_cpu_nthread(int cpu_nthread) {
    impl_->data.cpu_nthread = cpu_nthread;
  }

  int SystemConfiguration::get_cpu_nthread() const {
    return impl_->data.cpu_nthread;
  }

  void SystemConfiguration::set_gpu_device(int gpu_device) {
    impl_->data.gpu_device = gpu_device;
  }

  int SystemConfiguration::get_gpu_device() const {
    return impl_->data.gpu_device;
  }

  void SystemConfiguration::set_gpu_nstream(int gpu_nstream) {
    impl_->data.gpu_nstream = gpu_nstream;
  }

  int SystemConfiguration::get_gpu_nstream() const {
    return impl_->data.gpu_nstream;
  }

  void SystemConfiguration::set_nstream(int nstream) {
    impl_->data.nstream = nstream;
  }

  int SystemConfiguration::get_nstream() const {
    return impl_->data.nstream;
  }

  void SystemConfiguration::set_active(bool active) {
    impl_->data.active = active;
  }

  bool SystemConfiguration::get_active() const {
    return impl_->data.active;
  }

  bool SystemConfiguration::is_host() const {
    return impl_->data.is_host();
  }

  bool SystemConfiguration::is_device() const {
    return impl_->data.is_device();
  }

  bool SystemConfiguration::is_float() const {
    return impl_->data.is_float();
  }

  bool SystemConfiguration::is_double() const {
    return impl_->data.is_double();
  }

  bool SystemConfiguration::is_float_host() const {
    return impl_->data.is_float_host();
  }

  bool SystemConfiguration::is_double_host() const {
    return impl_->data.is_double_host();
  }

  bool SystemConfiguration::is_float_device() const {
    return impl_->data.is_float_device();
  }

  bool SystemConfiguration::is_double_device() const {
    return impl_->data.is_double_device();
  }
  
  /****************************************************************************
   * The AtomData interface
   ***************************************************************************/

  template <typename T>
  struct AtomData<T>::Data {
    /* Atom_Data<T> d_; */
    /* std::reference_wrapper<Atom_Data<T>> data; */
    Atom_Data<T> &data;
    Data(Atom_Data<T>& d)
      : data(d) {}
    Data(const Data &other)
      : data(other.data) {}
    Data& operator=(const Data &other) {
      data = other.data;
      return *this;
    }
    /* Data() */
    /*   : data(d_) {}; */
    /* Data(std::reference_wrapper<Atom_Data<T>> d) */
    /*   : data(d) {}; */
  };

  /* template <typename T> */
  /* AtomData<T>::AtomData() */
  /*     : impl_(std::make_unique<Data>()) {} */

  template <typename T>
  AtomData<T>::AtomData(const AtomData& other)
      : impl_(std::make_unique<Data>(*other.impl_)) {}

  template <typename T>
  AtomData<T>::AtomData(AtomData&& other) = default;

  template <typename T>
  AtomData<T>::AtomData(const AtomData<T>::Data& other)
      : impl_(std::make_unique<Data>(other)) {}

  template <typename T>
  AtomData<T>& AtomData<T>::operator=(const AtomData<T>& other) {
    *impl_ = *other.impl_;
    return *this;
  }

  template <typename T>
  AtomData<T>& AtomData<T>::operator=(AtomData<T>&&) = default;

  template <typename T>
  AtomData<T>::~AtomData<T>() = default;

  template <typename T>
  const AtomData<T>::Data& AtomData<T>::internal() const {
    return *impl_;
  }

  template <typename T>
  T AtomData<T>::get_dz() const {
    return impl_->data.dz;
  }

  template <typename T>
  void AtomData<T>::set_dz(T dz) {
    impl_->data.dz = dz;
  }
  
  template <typename T>
  T AtomData<T>::get_l_x() const {
    return impl_->data.l_x;
  }

  template <typename T>
  void AtomData<T>::set_l_x(T l_x) {
    impl_->data.l_x = l_x;
  }
  
  template <typename T>
  T AtomData<T>::get_l_y() const {
    return impl_->data.l_y;
  }

  template <typename T>
  void AtomData<T>::set_l_y(T l_y) {
    impl_->data.l_y = l_y;
  }
  
  template <typename T>
  T AtomData<T>::get_l_z() const {
    return impl_->data.l_z;
  }

  template <typename T>
  void AtomData<T>::set_l_z(T l_z) {
    impl_->data.l_z = l_z;
  }
  
  template <typename T>
  int AtomData<T>::get_ct_na() const {
    return impl_->data.ct_na;
  }

  template <typename T>
  void AtomData<T>::set_ct_na(int ct_na) {
    impl_->data.ct_na = ct_na;
  }
  
  template <typename T>
  int AtomData<T>::get_ct_nb() const {
    return impl_->data.ct_nb;
  }

  template <typename T>
  void AtomData<T>::set_ct_nb(int ct_nb) {
    impl_->data.ct_nb = ct_nb;
  }
  
  template <typename T>
  int AtomData<T>::get_ct_nc() const {
    return impl_->data.ct_nc;
  }

  template <typename T>
  void AtomData<T>::set_ct_nc(int ct_nc) {
    impl_->data.ct_nc = ct_nc;
  }
  
  template <typename T>
  T AtomData<T>::get_ct_a() const {
    return impl_->data.ct_a;
  }

  template <typename T>
  void AtomData<T>::set_ct_a(T ct_a) {
    impl_->data.ct_a = ct_a;
  }
  
  template <typename T>
  T AtomData<T>::get_ct_b() const {
    return impl_->data.ct_b;
  }

  template <typename T>
  void AtomData<T>::set_ct_b(T ct_b) {
    impl_->data.ct_b = ct_b;
  }
  
  template <typename T>
  T AtomData<T>::get_ct_c() const {
    return impl_->data.ct_c;
  }

  template <typename T>
  void AtomData<T>::set_ct_c(T ct_c) {
    impl_->data.ct_c = ct_c;
  }
  
  template <typename T>
  T AtomData<T>::get_ct_x0() const {
    return impl_->data.ct_x0;
  }

  template <typename T>
  void AtomData<T>::set_ct_x0(T ct_x0) {
    impl_->data.ct_x0 = ct_x0;
  }
  
  template <typename T>
  T AtomData<T>::get_ct_y0() const {
    return impl_->data.ct_y0;
  }

  template <typename T>
  void AtomData<T>::set_ct_y0(T ct_y0) {
    impl_->data.ct_y0 = ct_y0;
  }

  template <typename T>
  std::vector<Amorp_Lay_Info<T>> AtomData<T>::get_amorphous_parameters() const {
    return std::vector<Amorp_Lay_Info<T>>(
        impl_->data.amorp_lay_info.begin(),
        impl_->data.amorp_lay_info.end());
  }

  template <typename T>
  void AtomData<T>::set_amorphous_parameters(const std::vector<Amorp_Lay_Info<T>> &amorp_lay_info) {
    impl_->data.amorp_lay_info.assign(amorp_lay_info.begin(), amorp_lay_info.end());
  }
  
  template <typename T>
  std::vector<Atom<T>> AtomData<T>::get_spec_atoms() const {
    const Atom_Data<T>& atoms = impl_->data;
    std::vector<Atom<T>> result(atoms.size());
    for (auto i = 0; i < result.size(); ++i) {
      result[i] = Atom<T>(
          atoms.Z[i],
          atoms.x[i],
          atoms.y[i],
          atoms.z[i],
          atoms.sigma[i],
          atoms.occ[i],
          atoms.region[i],
          atoms.charge[i]);
    }
    return result;
  }

  template <typename T>
  void AtomData<T>::set_spec_atoms(const std::vector<Atom<T>>& spec_atoms) {
    Atom_Data<T>& atoms = impl_->data;
    atoms.resize(spec_atoms.size());
    for (auto i = 0; i < spec_atoms.size(); ++i) {
      atoms.Z[i] = spec_atoms[i].Z;
      atoms.x[i] = spec_atoms[i].x;
      atoms.y[i] = spec_atoms[i].y;
      atoms.z[i] = spec_atoms[i].z;
      atoms.sigma[i] = spec_atoms[i].sigma;
      atoms.occ[i] = spec_atoms[i].occ;
      atoms.region[i] = spec_atoms[i].region;
      atoms.charge[i] = spec_atoms[i].charge;
    }
  }

  template <typename T>
  void AtomData<T>::get_statistic() {
    impl_->data.get_statistic();
  }
  
  template <typename T>
  void AtomData<T>::clear() {
    impl_->data.clear();
  }
  
  template <typename T>
  bool AtomData<T>::empty() const {
    return impl_->data.empty();
  }
  
  template <typename T>
  AtomData<T>::size_type AtomData<T>::size() const {
    return impl_->data.size();
  }
    
  /****************************************************************************
   * The Scanning interface
   ***************************************************************************/

  template <typename T>
  struct ScanningData<T>::Data {
    Scanning<T>& data;
    Data(Scanning<T>& d)
      : data(d) {}
    Data(const Data &other)
      : data(other.data) {}
    Data& operator=(const Data &other) {
      data = other.data;
      return *this;
    }
  };

  template <typename T>
  ScanningData<T>::ScanningData(const ScanningData& other)
      : impl_(std::make_unique<Data>(*other.impl_)) {}

  template <typename T>
  ScanningData<T>::ScanningData(ScanningData&& other) = default;

  template <typename T>
  ScanningData<T>::ScanningData(const ScanningData<T>::Data& other)
      : impl_(std::make_unique<Data>(other)) {}

  template <typename T>
  ScanningData<T>& ScanningData<T>::operator=(const ScanningData<T>& other) {
    *impl_ = *other.impl_;
    return *this;
  }

  template <typename T>
  ScanningData<T>& ScanningData<T>::operator=(ScanningData<T>&&) = default;

  template <typename T>
  ScanningData<T>::~ScanningData<T>() = default;

  template <typename T>
  const ScanningData<T>::Data& ScanningData<T>::internal() const {
    return *impl_;
  }
    

  template <typename T>
  eScanning_Type ScanningData<T>::get_type() const {
    return impl_->data.type;
  }

  template <typename T>
  void ScanningData<T>::set_type(eScanning_Type type) {
    impl_->data.type = type;
  }
  

  template <typename T>
  bool ScanningData<T>::get_pbc() const {
    return impl_->data.pbc;
  }

  template <typename T>
  void ScanningData<T>::set_pbc(bool pbc) {
    impl_->data.pbc = pbc;
  }
  

  template <typename T>
  bool ScanningData<T>::get_spxs() const {
    return impl_->data.spxs;
  }

  template <typename T>
  void ScanningData<T>::set_spxs(bool spxs) {
    impl_->data.spxs = spxs;
  }
  

  template <typename T>
  int ScanningData<T>::get_ns() const {
    return impl_->data.ns;
  }

  template <typename T>
  void ScanningData<T>::set_ns(int ns) {
    impl_->data.ns = ns;
  }


  template <typename T>
  T ScanningData<T>::get_x0() const {
    return impl_->data.x0;
  }

  template <typename T>
  void ScanningData<T>::set_x0(T x0) {
    impl_->data.x0 = x0;
  }
  

  template <typename T>
  T ScanningData<T>::get_y0() const {
    return impl_->data.y0;
  }

  template <typename T>
  void ScanningData<T>::set_y0(T y0) {
    impl_->data.y0 = y0;
  }
  

  template <typename T>
  T ScanningData<T>::get_xe() const {
    return impl_->data.xe;
  }

  template <typename T>
  void ScanningData<T>::set_xe(T xe) {
    impl_->data.xe = xe;
  }
  

  template <typename T>
  T ScanningData<T>::get_ye() const {
    return impl_->data.ye;
  }

  template <typename T>
  void ScanningData<T>::set_ye(T ye) {
    impl_->data.ye = ye;
  }

  template <typename T>
  void ScanningData<T>::set_grid() {
    return impl_->data.set_grid();
  }
  
  /****************************************************************************
   * The Detector interface
   ***************************************************************************/

  template <typename T>
  struct DetectorData<T>::Data {
    Detector<T, e_host>& data;
    std::vector<T> inner_ang;
    std::vector<T> outer_ang;
    Data(Detector<T, e_host>& d)
      : data(d) {}
    Data(const Data &other)
      : data(other.data) {}
    Data& operator=(const Data &other) {
      data = other.data;
      return *this;
    }
  };

  template <typename T>
  DetectorData<T>::DetectorData(const DetectorData& other)
      : impl_(std::make_unique<Data>(*other.impl_)) {}

  template <typename T>
  DetectorData<T>::DetectorData(DetectorData&& other) = default;

  template <typename T>
  DetectorData<T>::DetectorData(const DetectorData<T>::Data& other)
      : impl_(std::make_unique<Data>(other)) {}

  template <typename T>
  DetectorData<T>& DetectorData<T>::operator=(const DetectorData<T>& other) {
    *impl_ = *other.impl_;
    return *this;
  }

  template <typename T>
  DetectorData<T>& DetectorData<T>::operator=(DetectorData<T>&&) = default;

  template <typename T>
  DetectorData<T>::~DetectorData<T>() = default;

  template <typename T>
  const DetectorData<T>::Data& DetectorData<T>::internal() const {
    return *impl_;
  }
		
  template <typename T>
  DetectorData<T>::size_type DetectorData<T>::size() const {
    return impl_->data.size();
  }

  template <typename T>
  void DetectorData<T>::clear() {
    impl_->data.clear();
  }

  template <typename T>
  void DetectorData<T>::resize(const typename DetectorData<T>::size_type &new_size) {
    impl_->data.resize(new_size);
  }
  
  template <typename T>
  bool DetectorData<T>::is_detector_circular() const {
    return impl_->data.is_detector_circular();
  }

  template <typename T>
  bool DetectorData<T>::is_detector_radial() const {
    return impl_->data.is_detector_radial();
  }

  template <typename T>
  bool DetectorData<T>::is_detector_matrix() const {
    return impl_->data.is_detector_matrix();
  }

  
  template <typename T>
  eDetector_Type DetectorData<T>::get_type() const {
    return impl_->data.type;
  }

  template <typename T>
  void DetectorData<T>::set_type(eDetector_Type type) {
    impl_->data.type = type;
  }

  template <typename T>
  std::vector<T> DetectorData<T>::get_g_inner() const {
    return std::vector<T>(impl_->data.g_inner.begin(), impl_->data.g_inner.end());
  }

  template <typename T>
  void DetectorData<T>::set_g_inner(const std::vector<T>& g_inner) {
    impl_->data.g_inner.assign(g_inner.begin(), g_inner.end());
  }

  template <typename T>
  std::vector<T> DetectorData<T>::get_g_outer() const {
    return std::vector<T>(impl_->data.g_outer.begin(), impl_->data.g_outer.end());
  }

  template <typename T>
  void DetectorData<T>::set_g_outer(const std::vector<T>& g_outer) {
    impl_->data.g_outer.assign(g_outer.begin(), g_outer.end());
  }

  template <typename T>
  std::vector<std::vector<T>> DetectorData<T>::get_fx() const {
    std::vector<std::vector<T>> result;
    for (auto x : impl_->data.fx) {
      result.push_back(std::vector<T>(x.begin(), x.end()));
    }
    return result;
  }

  template <typename T>
  void DetectorData<T>::set_fx(const std::vector<std::vector<T>>& fx) {
    Vector<Vector<T, e_host>, e_host> result;
    for (auto x : fx) {
      result.push_back(Vector<T, e_host>(x.begin(), x.end()));
    }
    impl_->data.fx = result;
  }
  
  template <typename T>
  std::vector<std::vector<T>> DetectorData<T>::get_fR() const {
    std::vector<std::vector<T>> result;
    for (auto x : impl_->data.fR) {
      result.push_back(std::vector<T>(x.begin(), x.end()));
    }
    return result;
  }

  template <typename T>
  void DetectorData<T>::set_fR(const std::vector<std::vector<T>>& fR) {
    Vector<Vector<T, e_host>, e_host> result;
    for (auto x : fR) {
      result.push_back(Vector<T, e_host>(x.begin(), x.end()));
    }
    impl_->data.fR = result;
  }
  
  template <typename T>
  std::vector<Grid_2d<T>> DetectorData<T>::get_grid_1d() const {
    return impl_->data.grid_1d;
  }

  template <typename T>
  void DetectorData<T>::set_grid_1d(const std::vector<Grid_2d<T>>& grid_1d) {
    impl_->data.grid_1d = grid_1d;
  }
  
  template <typename T>
  std::vector<Grid_2d<T>> DetectorData<T>::get_grid_2d() const {
    return impl_->data.grid_2d;
  }

  template <typename T>
  void DetectorData<T>::set_grid_2d(const std::vector<Grid_2d<T>>& grid_2d) {
    impl_->data.grid_2d = grid_2d;
  }
  
  template <typename T>
  std::vector<std::string> DetectorData<T>::get_fn() const {
    return impl_->data.fn;
  }

  template <typename T>
  void DetectorData<T>::set_fn(const std::vector<std::string>& fn) {
    impl_->data.fn = fn;
  }

  template <typename T>
  std::vector<T> DetectorData<T>::get_inner_ang() const {
    return std::vector<T>(impl_->inner_ang.begin(), impl_->inner_ang.end());
  }

  template <typename T>
  void DetectorData<T>::set_inner_ang(const std::vector<T>& inner_ang) {
    impl_->inner_ang.assign(inner_ang.begin(), inner_ang.end());
  }

  template <typename T>
  std::vector<T> DetectorData<T>::get_outer_ang() const {
    return std::vector<T>(impl_->outer_ang.begin(), impl_->outer_ang.end());
  }

  template <typename T>
  void DetectorData<T>::set_outer_ang(const std::vector<T>& outer_ang) {
    impl_->outer_ang.assign(outer_ang.begin(), outer_ang.end());
  }

  /****************************************************************************
   * The Input interface
   ***************************************************************************/

  template <typename T>
  struct Input<T>::Data {
    Input_Multislice<T> data;
    AtomData<T> atom_data_proxy;
    ScanningData<T> scanning_proxy;
    DetectorData<T> detector_proxy;
    Data()
      : atom_data_proxy(typename AtomData<T>::Data(data.atoms)),
        scanning_proxy(typename ScanningData<T>::Data(data.scanning)),
        detector_proxy(typename DetectorData<T>::Data(data.detector)){}
  };

  template <typename T>
  Input<T>::Input()
      : impl_(std::make_unique<Data>()) {}

  template <typename T>
  Input<T>::Input(const Input& other)
      : impl_(std::make_unique<Data>(*other.impl_)) {}

  template <typename T>
  Input<T>::Input(Input&& other) = default;

  template <typename T>
  Input<T>::Input(const Input<T>::Data& other)
      : impl_(std::make_unique<Data>(other)) {}

  template <typename T>
  Input<T>& Input<T>::operator=(const Input<T>& other) {
    *impl_ = *other.impl_;
    return *this;
  }

  template <typename T>
  Input<T>& Input<T>::operator=(Input<T>&&) = default;

  template <typename T>
  Input<T>::~Input<T>() = default;

  template <typename T>
  Input<T>::Data& Input<T>::internal() {
    return *impl_;
  }

  template <typename T>
  SystemConfiguration Input<T>::get_system_conf() const {
    return SystemConfiguration(SystemConfiguration::Data(impl_->data.system_conf));
  }

  template <typename T>
  void Input<T>::set_system_conf(const SystemConfiguration& system_conf) {
    impl_->data.system_conf = system_conf.internal().data;
  }

  template <typename T>
  eElec_Spec_Int_Model Input<T>::get_interaction_model() const {
    return impl_->data.interaction_model;
  }

  template <typename T>
  void Input<T>::set_interaction_model(eElec_Spec_Int_Model interaction_model) {
    impl_->data.interaction_model = interaction_model;
  }

  template <typename T>
  ePotential_Type Input<T>::get_potential_type() const {
    return impl_->data.potential_type;
  }

  template <typename T>
  void Input<T>::set_potential_type(ePotential_Type potential_type) {
    impl_->data.potential_type = potential_type;
  }

  template <typename T>
  ePhonon_Model Input<T>::get_pn_model() const {
    return impl_->data.pn_model;
  }

  template <typename T>
  void Input<T>::set_pn_model(ePhonon_Model pn_model) {
    impl_->data.pn_model = pn_model;
  }

  template <typename T>
  bool Input<T>::get_pn_coh_contrib() const {
    return impl_->data.pn_coh_contrib;
  }

  template <typename T>
  void Input<T>::set_pn_coh_contrib(bool pn_coh_contrib) {
    impl_->data.pn_coh_contrib = pn_coh_contrib;
  }

  template <typename T>
  bool Input<T>::get_pn_single_conf() const {
    return impl_->data.pn_single_conf;
  }

  template <typename T>
  void Input<T>::set_pn_single_conf(bool pn_single_conf) {
    impl_->data.pn_single_conf = pn_single_conf;
  }

  template <typename T>
  FP_Dim& Input<T>::get_pn_dim() const {
    return impl_->data.pn_dim;
  }

  template <typename T>
  void Input<T>::set_pn_dim(const FP_Dim &pn_dim) {
    impl_->data.pn_dim = pn_dim;
  }

  template <typename T>
  int Input<T>::get_fp_dist() const {
    return impl_->data.fp_dist;
  }

  template <typename T>
  void Input<T>::set_fp_dist(int fp_dist) {
    impl_->data.fp_dist = fp_dist;
  }

  template <typename T>
  int Input<T>::get_pn_seed() const {
    return impl_->data.pn_seed;
  }

  template <typename T>
  void Input<T>::set_pn_seed(int pn_seed) {
    impl_->data.pn_seed = pn_seed;
  }

  template <typename T>
  int Input<T>::get_pn_nconf() const {
    return impl_->data.pn_nconf;
  }

  template <typename T>
  void Input<T>::set_pn_nconf(int pn_nconf) {
    impl_->data.pn_nconf = pn_nconf;
  }

  template <typename T>
  int Input<T>::get_fp_iconf_0() const {
    return impl_->data.fp_iconf_0;
  }

  template <typename T>
  void Input<T>::set_fp_iconf_0(int fp_iconf_0) {
    impl_->data.fp_iconf_0 = fp_iconf_0;
  }

  template <typename T>
  AtomData<T>& Input<T>::get_atoms() const {
    return impl_->atom_data_proxy;
  }

  template <typename T>
  void Input<T>::set_atoms(const AtomData<T>& atoms) {
    impl_->data.atoms = atoms.internal().data;
    impl_->atom_data_proxy = AtomData<T>(typename AtomData<T>::Data(impl_->data.atoms));
  }

  template <typename T>
  bool Input<T>::get_is_crystal() const {
    return impl_->data.is_crystal;
  }

  template <typename T>
  void Input<T>::set_is_crystal(bool is_crystal) {
    impl_->data.is_crystal = is_crystal;
  }

  template <typename T>
  double Input<T>::get_spec_rot_theta() const {
    return impl_->data.spec_rot_theta;
  }

  template <typename T>
  void Input<T>::set_spec_rot_theta(double spec_rot_theta) {
    impl_->data.spec_rot_theta = spec_rot_theta;
  }

  template <typename T>
  r3d<T>& Input<T>::get_spec_rot_u0() const {
    return impl_->data.spec_rot_u0;
  }

  template <typename T>
  void Input<T>::set_spec_rot_u0(const r3d<T>& spec_rot_u0) {
    impl_->data.spec_rot_u0 = spec_rot_u0;
  }

  template <typename T>
  eRot_Point_Type Input<T>::get_spec_rot_center_type() const {
    return impl_->data.spec_rot_center_type;
  }

  template <typename T>
  void Input<T>::set_spec_rot_center_type(eRot_Point_Type spec_rot_center_type) {
    impl_->data.spec_rot_center_type = spec_rot_center_type;
  }

  template <typename T>
  r3d<T>& Input<T>::get_spec_rot_center_p() const {
    return impl_->data.spec_rot_center_p;
  }

  template <typename T>
  void Input<T>::set_spec_rot_center_p(const r3d<T>& spec_rot_center_p) {
    impl_->data.spec_rot_center_p = spec_rot_center_p;
  }

  template <typename T>
  eThick_Type Input<T>::get_thick_type() const {
    return impl_->data.thick_type;
  }

  template <typename T>
  void Input<T>::set_thick_type(eThick_Type thick_type) {
    impl_->data.thick_type = thick_type;
  }

  template <typename T>
  std::vector<T> Input<T>::get_thick() const {
    return std::vector<T>(impl_->data.thick.begin(), impl_->data.thick.end());
  }

  template <typename T>
  void Input<T>::set_thick(const std::vector<T>& thick) {
    impl_->data.thick.assign(thick.begin(), thick.end());
  }

  template <typename T>
  ePotential_Slicing Input<T>::get_potential_slicing() const {
    return impl_->data.potential_slicing;
  }

  template <typename T>
  void Input<T>::set_potential_slicing(ePotential_Slicing potential_slicing) {
    impl_->data.potential_slicing = potential_slicing;
  }

  template <typename T>
  Grid_2d<T>& Input<T>::get_grid_2d() const {
    return impl_->data.grid_2d;
  }

  template <typename T>
  void Input<T>::set_grid_2d(const Grid_2d<T>& grid_2d) {
    impl_->data.grid_2d = grid_2d;
  }

  template <typename T>
  Range_2d& Input<T>::get_output_area() const {
    return impl_->data.output_area;
  }

  template <typename T>
  void Input<T>::set_output_area(const Range_2d& output_area) {
    impl_->data.output_area = output_area;
  }

  template <typename T>
  eTEM_Sim_Type Input<T>::get_simulation_type() const {
    return impl_->data.simulation_type;
  }

  template <typename T>
  void Input<T>::set_simulation_type(eTEM_Sim_Type simulation_type) {
    impl_->data.simulation_type = simulation_type;
  }

  template <typename T>
  eIncident_Wave_Type Input<T>::get_iw_type() const {
    return impl_->data.iw_type;
  }

  template <typename T>
  void Input<T>::set_iw_type(eIncident_Wave_Type iw_type) {
    impl_->data.iw_type = iw_type;
  }

  template <typename T>
  std::vector<std::complex<T>> Input<T>::get_iw_psi() const {
    return std::vector<std::complex<T>>(impl_->data.iw_psi.begin(), impl_->data.iw_psi.end());
  }

  template <typename T>
  void Input<T>::set_iw_psi(const std::vector<std::complex<T>>& iw_psi) {
    impl_->data.iw_psi.assign(iw_psi.begin(), iw_psi.end());
  }

  template <typename T>
  std::vector<T> Input<T>::get_iw_x() const {
    return std::vector<T>(impl_->data.iw_x.begin(), impl_->data.iw_x.end());
  }

  template <typename T>
  void Input<T>::set_iw_x(const std::vector<T>& iw_x) {
    impl_->data.iw_x.assign(iw_x.begin(), iw_x.end());
  }

  template <typename T>
  std::vector<T> Input<T>::get_iw_y() const {
    return std::vector<T>(impl_->data.iw_y.begin(), impl_->data.iw_y.end());
  }

  template <typename T>
  void Input<T>::set_iw_y(const std::vector<T>& iw_y) {
    impl_->data.iw_y.assign(iw_y.begin(), iw_y.end());
  }

  template <typename T>
  double Input<T>::get_E_0() const {
    return impl_->data.E_0;
  }

  template <typename T>
  void Input<T>::set_E_0(double E_0) {
    impl_->data.E_0 = E_0;
  }

  template <typename T>
  double Input<T>::get_lambda() const {
    return impl_->data.lambda;
  }

  template <typename T>
  void Input<T>::set_lambda(double lambda) {
    impl_->data.lambda = lambda;
  }

  template <typename T>
  double Input<T>::get_theta() const {
    return impl_->data.theta;
  }

  template <typename T>
  void Input<T>::set_theta(double theta) {
    impl_->data.theta = theta;
  }

  template <typename T>
  double Input<T>::get_phi() const {
    return impl_->data.phi;
  }

  template <typename T>
  void Input<T>::set_phi(double phi) {
    impl_->data.phi = phi;
  }

  template <typename T>
  eIllumination_Model Input<T>::get_illumination_model() const {
    return impl_->data.illumination_model;
  }

  template <typename T>
  void Input<T>::set_illumination_model(eIllumination_Model illumination_model) {
    impl_->data.illumination_model = illumination_model;
  }

  template <typename T>
  eTemporal_Spatial_Incoh Input<T>::get_temporal_spatial_incoh() const {
    return impl_->data.temporal_spatial_incoh;
  }

  template <typename T>
  void Input<T>::set_temporal_spatial_incoh(eTemporal_Spatial_Incoh temporal_spatial_incoh) {
    impl_->data.temporal_spatial_incoh = temporal_spatial_incoh;
  }

  template <typename T>
  Lens<T>& Input<T>::get_cond_lens() const {
    return impl_->data.cond_lens;
  }
  
  template <typename T>
  void Input<T>::set_cond_lens(const Lens<T>& cond_lens) {
    impl_->data.cond_lens = cond_lens;
  }

  template <typename T>
  Lens<T>& Input<T>::get_obj_lens() const {
    return impl_->data.obj_lens;
  }
  
  template <typename T>
  void Input<T>::set_obj_lens(const Lens<T>& obj_lens) {
    impl_->data.obj_lens = obj_lens;
  }

  template <typename T>
  ScanningData<T>& Input<T>::get_scanning() const {
    return impl_->scanning_proxy;
  }
  
  template <typename T>
  void Input<T>::set_scanning(const ScanningData<T>& scanning) {
    impl_->data.scanning = scanning.internal().data;
    impl_->scanning_proxy = ScanningData<T>(typename ScanningData<T>::Data(impl_->data.scanning));
  }

  template <typename T>
  DetectorData<T>& Input<T>::get_detector() const {
    return impl_->detector_proxy;
  }
  
  template <typename T>
  void Input<T>::set_detector(const DetectorData<T>& detector) {
    impl_->data.detector = detector.internal().data;
    impl_->detector_proxy = DetectorData<T>(typename DetectorData<T>::Data(impl_->data.detector));
  }

  template <typename T>
  EELS<T>& Input<T>::get_eels_fr() const {
    return impl_->data.eels_fr;
  }
  
  template <typename T>
  void Input<T>::set_eels_fr(const EELS<T>& eels_fr) {
    impl_->data.eels_fr = eels_fr;
  }

  template <typename T>
  eOperation_Mode Input<T>::get_operation_mode() const {
    return impl_->data.operation_mode;
  }

  template <typename T>
  void Input<T>::set_operation_mode(eOperation_Mode operation_mode) {
    impl_->data.operation_mode = operation_mode;
  }

  template <typename T>
  bool Input<T>::get_slice_storage() const {
    return impl_->data.slice_storage;
  }

  template <typename T>
  void Input<T>::set_slice_storage(bool slice_storage) {
    impl_->data.slice_storage = slice_storage;
  }

  template <typename T>
  bool Input<T>::get_reverse_multislice() const {
    return impl_->data.reverse_multislice;
  }

  template <typename T>
  void Input<T>::set_reverse_multislice(bool reverse_multislice) {
    impl_->data.reverse_multislice = reverse_multislice;
  }

  template <typename T>
  int Input<T>::get_mul_sign() const {
    return impl_->data.mul_sign;
  }

  template <typename T>
  void Input<T>::set_mul_sign(int mul_sign) {
    impl_->data.mul_sign = mul_sign;
  }

  template <typename T>
  double Input<T>::get_Vrl() const {
    return impl_->data.Vrl;
  }

  template <typename T>
  void Input<T>::set_Vrl(double Vrl) {
    impl_->data.Vrl = Vrl;
  }

  template <typename T>
  int Input<T>::get_nR() const {
    return impl_->data.nR;
  }

  template <typename T>
  void Input<T>::set_nR(int nR) {
    impl_->data.nR = nR;
  }

  template <typename T>
  int Input<T>::get_nrot() const {
    return impl_->data.nrot;
  }

  template <typename T>
  void Input<T>::set_nrot(int nrot) {
    impl_->data.nrot = nrot;
  }

  template <typename T>
  eLens_Var_Type Input<T>::get_cdl_var_type() const {
    return impl_->data.cdl_var_type;
  }

  template <typename T>
  void Input<T>::set_cdl_var_type(eLens_Var_Type cdl_var_type) {
    impl_->data.cdl_var_type = cdl_var_type;
  }

  template <typename T>
  std::vector<T> Input<T>::get_cdl_var() const {
    return std::vector<T>(impl_->data.cdl_var.begin(), impl_->data.cdl_var.end());
  }

  template <typename T>
  void Input<T>::set_cdl_var(const std::vector<T>& cdl_var) {
    impl_->data.cdl_var.assign(cdl_var.begin(), cdl_var.end());
  }

  template <typename T>
  std::vector<int> Input<T>::get_iscan() const {
    return std::vector<int>(impl_->data.iscan.begin(), impl_->data.iscan.end());
  }

  template <typename T>
  void Input<T>::set_iscan(const std::vector<int>& iscan) {
    impl_->data.iscan.assign(iscan.begin(), iscan.end());
  }

  template <typename T>
  std::vector<T> Input<T>::get_beam_x() const {
    return std::vector<T>(impl_->data.beam_x.begin(), impl_->data.beam_x.end());
  }

  template <typename T>
  void Input<T>::set_beam_x(const std::vector<T>& beam_x) {
    impl_->data.beam_x.assign(beam_x.begin(), beam_x.end());
  }

  template <typename T>
  std::vector<T> Input<T>::get_beam_y() const {
    return std::vector<T>(impl_->data.beam_y.begin(), impl_->data.beam_y.end());
  }

  template <typename T>
  void Input<T>::set_beam_y(const std::vector<T>& beam_y) {
    impl_->data.beam_y.assign(beam_y.begin(), beam_y.end());
  }

  template <typename T>
  int Input<T>::get_islice() const {
    return impl_->data.islice;
  }

  template <typename T>
  void Input<T>::set_islice(int islice) {
    impl_->data.islice = islice;
  }

  template <typename T>
  bool Input<T>::get_dp_Shift() const {
    return impl_->data.dp_Shift;
  }

  template <typename T>
  void Input<T>::set_dp_Shift(bool dp_Shift) {
    impl_->data.dp_Shift = dp_Shift;
  }
  
  template <typename T>
  void Input<T>::set_incident_wave_type(eIncident_Wave_Type iw_type) {
    impl_->data.set_incident_wave_type(iw_type);
  }

  template <typename T>
  void Input<T>::validate_parameters() {
    impl_->data.validate_parameters();
  }

  template <typename T>
  bool Input<T>::is_multislice() const {
    return impl_->data.is_multislice();
  }

  template <typename T>
  bool Input<T>::is_phase_object() const {
    return impl_->data.is_phase_object();
  }

  template <typename T>
  bool Input<T>::is_weak_phase_object() const {
    return impl_->data.is_weak_phase_object();
  }

  template <typename T>
  bool Input<T>::is_still_atom() const {
    return impl_->data.is_still_atom();
  }

  template <typename T>
  bool Input<T>::is_absorptive_model() const {
    return impl_->data.is_absorptive_model();
  }

  template <typename T>
  bool Input<T>::is_frozen_phonon() const {
    return impl_->data.is_frozen_phonon();
  }

  template <typename T>
  bool Input<T>::is_frozen_phonon_single_conf() const {
    return impl_->data.is_frozen_phonon_single_conf();
  }

  template <typename T>
  bool Input<T>::is_whole_spec() const {
    return impl_->data.is_whole_spec();
  }

  template <typename T>
  bool Input<T>::is_through_slices() const {
    return impl_->data.is_through_slices();
  }

  template <typename T>
  bool Input<T>::is_through_thick() const {
    return impl_->data.is_through_thick();
  }

  template <typename T>
  bool Input<T>::is_slicing_by_planes() const {
    return impl_->data.is_slicing_by_planes();
  }

  template <typename T>
  bool Input<T>::is_slicing_by_dz() const {
    return impl_->data.is_slicing_by_dz();
  }

  template <typename T>
  bool Input<T>::is_subslicing() const {
    return impl_->data.is_subslicing();
  }

  template <typename T>
  bool Input<T>::is_subslicing_whole_spec() const {
    return impl_->data.is_subslicing_whole_spec();
  }

  template <typename T>
  bool Input<T>::is_plane_wave() const {
    return impl_->data.is_plane_wave();
  }

  template <typename T>
  bool Input<T>::is_convergent_wave() const {
    return impl_->data.is_convergent_wave();
  }

  template <typename T>
  bool Input<T>::is_user_define_wave() const {
    return impl_->data.is_user_define_wave();
  }

  template <typename T>
  bool Input<T>::is_STEM() const {
    return impl_->data.is_STEM();
  }

  template <typename T>
  bool Input<T>::is_ISTEM() const {
    return impl_->data.is_ISTEM();
  }

  template <typename T>
  bool Input<T>::is_CBED() const {
    return impl_->data.is_CBED();
  }

  template <typename T>
  bool Input<T>::is_CBEI() const {
    return impl_->data.is_CBEI();
  }

  template <typename T>
  bool Input<T>::is_ED() const {
    return impl_->data.is_ED();
  }

  template <typename T>
  bool Input<T>::is_HRTEM() const {
    return impl_->data.is_HRTEM();
  }

  template <typename T>
  bool Input<T>::is_PED() const {
    return impl_->data.is_PED();
  }

  template <typename T>
  bool Input<T>::is_HCTEM() const {
    return impl_->data.is_HCTEM();
  }

  template <typename T>
  bool Input<T>::is_EWFS() const {
    return impl_->data.is_EWFS();
  }

  template <typename T>
  bool Input<T>::is_EWRS() const {
    return impl_->data.is_EWRS();
  }

  template <typename T>
  bool Input<T>::is_EWFS_SC() const {
    return impl_->data.is_EWFS_SC();
  }

  template <typename T>
  bool Input<T>::is_EWRS_SC() const {
    return impl_->data.is_EWRS_SC();
  }

  template <typename T>
  bool Input<T>::is_EELS() const {
    return impl_->data.is_EELS();
  }

  template <typename T>
  bool Input<T>::is_EFTEM() const {
    return impl_->data.is_EFTEM();
  }

  template <typename T>
  bool Input<T>::is_IWFS() const {
    return impl_->data.is_IWFS();
  }

  template <typename T>
  bool Input<T>::is_IWRS() const {
    return impl_->data.is_IWRS();
  }

  template <typename T>
  bool Input<T>::is_PPFS() const {
    return impl_->data.is_PPFS();
  }

  template <typename T>
  bool Input<T>::is_PPRS() const {
    return impl_->data.is_PPRS();
  }

  template <typename T>
  bool Input<T>::is_TFFS() const {
    return impl_->data.is_TFFS();
  }

  template <typename T>
  bool Input<T>::is_TFRS() const {
    return impl_->data.is_TFRS();
  }

  template <typename T>
  bool Input<T>::is_PropFS() const {
    return impl_->data.is_PropFS();
  }

  template <typename T>
  bool Input<T>::is_PropRS() const {
    return impl_->data.is_PropRS();
  }

  template <typename T>
  bool Input<T>::is_STEM_ISTEM() const {
    return impl_->data.is_STEM_ISTEM();
  }

  template <typename T>
  bool Input<T>::is_CBED_CBEI() const {
    return impl_->data.is_CBED_CBEI();
  }

  template <typename T>
  bool Input<T>::is_ED_HRTEM() const {
    return impl_->data.is_ED_HRTEM();
  }

  template <typename T>
  bool Input<T>::is_PED_HCTEM() const {
    return impl_->data.is_PED_HCTEM();
  }

  template <typename T>
  bool Input<T>::is_EWFS_EWRS() const {
    return impl_->data.is_EWFS_EWRS();
  }

  template <typename T>
  bool Input<T>::is_EWFS_EWRS_SC() const {
    return impl_->data.is_EWFS_EWRS_SC();
  }

  template <typename T>
  bool Input<T>::is_EELS_EFTEM() const {
    return impl_->data.is_EELS_EFTEM();
  }

  template <typename T>
  bool Input<T>::is_IWFS_IWRS() const {
    return impl_->data.is_IWFS_IWRS();
  }

  template <typename T>
  bool Input<T>::is_PPFS_PPRS() const {
    return impl_->data.is_PPFS_PPRS();
  }

  template <typename T>
  bool Input<T>::is_TFFS_TFRS() const {
    return impl_->data.is_TFFS_TFRS();
  }

  template <typename T>
  bool Input<T>::is_PropFS_PropRS() const {
    return impl_->data.is_PropFS_PropRS();
  }

  template <typename T>
  bool Input<T>::is_grid_FS() const {
    return impl_->data.is_grid_FS();
  }

  template <typename T>
  bool Input<T>::is_grid_RS() const {
    return impl_->data.is_grid_RS();
  }

  template <typename T>
  bool Input<T>::is_simulation_type_FS() const {
    return impl_->data.is_simulation_type_FS();
  }

  template <typename T>
  bool Input<T>::is_simulation_type_RS() const {
    return impl_->data.is_simulation_type_RS();
  }

  template <typename T>
  bool Input<T>::is_specimen_required() const {
    return impl_->data.is_specimen_required();
  }

  template <typename T>
  bool Input<T>::is_ISTEM_CBEI_HRTEM_HCTEM_EFTEM() const {
    return impl_->data.is_ISTEM_CBEI_HRTEM_HCTEM_EFTEM();
  }

  template <typename T>
  bool Input<T>::is_CBED_ED_EWFS_PED() const {
    return impl_->data.is_CBED_ED_EWFS_PED();
  }

  template <typename T>
  bool Input<T>::is_obj_lens_temp_spat() const {
    return impl_->data.is_obj_lens_temp_spat();
  }

  template <typename T>
  bool Input<T>::is_cond_lens_temp_spat() const {
    return impl_->data.is_cond_lens_temp_spat();
  }

  template <typename T>
  bool Input<T>::is_scanning() const {
    return impl_->data.is_scanning();
  }

  template <typename T>
  bool Input<T>::is_illu_mod_coherent() const {
    return impl_->data.is_illu_mod_coherent();
  }

  template <typename T>
  bool Input<T>::is_illu_mod_partial_coherent() const {
    return impl_->data.is_illu_mod_partial_coherent();
  }

  template <typename T>
  bool Input<T>::is_illu_mod_trans_cross_coef() const {
    return impl_->data.is_illu_mod_trans_cross_coef();
  }

  template <typename T>
  bool Input<T>::is_illu_mod_full_integration() const {
    return impl_->data.is_illu_mod_full_integration();
  }

  template <typename T>
  bool Input<T>::is_incoh_temporal_spatial() const {
    return impl_->data.is_incoh_temporal_spatial();
  }

  template <typename T>
  bool Input<T>::is_incoh_temporal() const {
    return impl_->data.is_incoh_temporal();
  }

  template <typename T>
  bool Input<T>::is_incoh_spatial() const {
    return impl_->data.is_incoh_spatial();
  }

  template <typename T>
  bool Input<T>::is_detector_circular() const {
    return impl_->data.is_detector_circular();
  }

  template <typename T>
  bool Input<T>::is_detector_radial() const {
    return impl_->data.is_detector_radial();
  }

  template <typename T>
  bool Input<T>::is_detector_matrix() const {
    return impl_->data.is_detector_matrix();
  }

  template <typename T>
  bool Input<T>::is_slice_storage() const {
    return impl_->data.is_slice_storage();
  }

  template <typename T>
  bool Input<T>::is_operation_mode_normal() const {
    return impl_->data.is_operation_mode_normal();
  }

  template <typename T>
  bool Input<T>::is_operation_mode_advanced() const {
    return impl_->data.is_operation_mode_advanced();
  }

  template <typename T>
  bool Input<T>::is_lvt_off() const {
    return impl_->data.is_lvt_off();
  }

  template <typename T>
  bool Input<T>::is_lvt_m() const {
    return impl_->data.is_lvt_m();
  }

  template <typename T>
  bool Input<T>::is_lvt_Cs3() const {
    return impl_->data.is_lvt_Cs3();
  }

  template <typename T>
  bool Input<T>::is_lvt_Cs5() const {
    return impl_->data.is_lvt_Cs5();
  }

  template <typename T>
  bool Input<T>::is_lvt_mfa2() const {
    return impl_->data.is_lvt_mfa2();
  }

  template <typename T>
  bool Input<T>::is_lvt_afa2() const {
    return impl_->data.is_lvt_afa2();
  }

  template <typename T>
  bool Input<T>::is_lvt_mfa3() const {
    return impl_->data.is_lvt_mfa3();
  }

  template <typename T>
  bool Input<T>::is_lvt_afa3() const {
    return impl_->data.is_lvt_afa3();
  }

  template <typename T>
  bool Input<T>::is_lvt_inner_aper_ang() const {
    return impl_->data.is_lvt_inner_aper_ang();
  }

  template <typename T>
  bool Input<T>::is_lvt_outer_aper_ang() const {
    return impl_->data.is_lvt_outer_aper_ang();
  }

  /****************************************************************************
   * The Output interface
   ***************************************************************************/

  template <typename T>
  struct Output<T>::Data {
    Output_Multislice<T> data;

    // Constructors required due to behaviour of Output_Multislice. Would
    // probably be better if we could fix Output_Multislice so that the normal
    // copy constructors work.
    Data() = default;
    Data(Data&) = default;
    Data& operator=(Data& other) {
      data = other.data;
      return *this;
    }
  };

  template <typename T>
  Output<T>::Output()
      : impl_(std::make_unique<Data>()) {}

  template <typename T>
  Output<T>::Output(Output& other)
      : impl_(std::make_unique<Data>(*other.impl_)) {}

  template <typename T>
  Output<T>::Output(Output&& other) = default;

  template <typename T>
  Output<T>::Output(Output<T>::Data& other)
      : impl_(std::make_unique<Data>(other)) {}

  template <typename T>
  Output<T>& Output<T>::operator=(Output<T>& other) {
    *impl_ = *other.impl_;
    return *this;
  }

  template <typename T>
  Output<T>& Output<T>::operator=(Output<T>&&) = default;

  template <typename T>
  Output<T>::~Output<T>() = default;

  template <typename T>
  const Output<T>::Data& Output<T>::internal() const {
    return *impl_;
  }
  
  template <typename T>
  Output<T>::Data& Output<T>::internal() {
    return *impl_;
  }

  template <typename T>
  void Output<T>::set_input_data(Input<T> &input) {
    impl_->data.set_input_data(&input.internal().data);
  }

  template <typename T>
  eTEM_Output_Type Output<T>::get_output_type() const {
    return impl_->data.output_type;
  }
  
  template <typename T>
  void Output<T>::set_output_type(eTEM_Output_Type output_type) {
    impl_->data.output_type = output_type;
  }

  template <typename T>
  int Output<T>::get_ndetector() const {
    return impl_->data.ndetector;
  }
		
  template <typename T>
  void Output<T>::set_ndetector(int ndetector) {
    impl_->data.ndetector = ndetector;
  }

  template <typename T>
  int Output<T>::get_nx() const {
    return impl_->data.nx;
  }

  template <typename T>
  void Output<T>::set_nx(int nx) {
    impl_->data.nx = nx;
  }

  template <typename T>
  int Output<T>::get_ny() const {
    return impl_->data.ny;
  }

  template <typename T>
  void Output<T>::set_ny(int ny) {
    impl_->data.ny = ny;
  }

  template <typename T>
  T Output<T>::get_dx() const {
    return impl_->data.dx;
  }

  template <typename T>
  void Output<T>::set_dx(T dx) {
    impl_->data.dx = dx;
  }

  template <typename T>
  T Output<T>::get_dy() const {
    return impl_->data.dy;
  }

  template <typename T>
  void Output<T>::set_dy(T dy) {
    impl_->data.dy = dy;
  }

  template <typename T>
  T Output<T>::get_dr() const {
    return impl_->data.dr;
  }

  template <typename T>
  void Output<T>::set_dr(T dr) {
    impl_->data.dr = dr;
  }

  template <typename T>
  std::vector<T> Output<T>::get_x() const {
    return std::vector<T>(impl_->data.x.begin(), impl_->data.x.end());
  }

  template <typename T>
  void Output<T>::set_x(const std::vector<T>& x) {
    impl_->data.x.assign(x.begin(), x.end());
  }

  template <typename T>
  std::vector<T> Output<T>::get_y() const {
    return std::vector<T>(impl_->data.y.begin(), impl_->data.y.end());
  }

  template <typename T>
  void Output<T>::set_y(const std::vector<T>& y) {
    impl_->data.y.assign(y.begin(), y.end());
  }

  template <typename T>
  std::vector<T> Output<T>::get_r() const {
    return std::vector<T>(impl_->data.r.begin(), impl_->data.r.end());
  }

  template <typename T>
  void Output<T>::set_r(const std::vector<T>& r) {
    impl_->data.r.assign(r.begin(), r.end());
  }
    
  template <typename T>
  std::vector<DetInt<typename Output<T>::vector_type>> Output<T>::get_image_tot() const {
    std::vector<DetInt<vector_type>> result;
    for (auto &x : impl_->data.image_tot) {
      DetInt<vector_type> d;
      for (auto &y : x.image) {
        d.image.push_back(vector_type(y.begin(), y.end()));
      }
      result.push_back(d);
    }
    return result;
  }

  template <typename T>
  void Output<T>::set_image_tot(const std::vector<DetInt<vector_type>>& image_tot) {
    typedef typename Output_Multislice<T>::TVector_hr TVector_hr;
    host_vector<Det_Int<TVector_hr>> result;
    for (auto &x : image_tot) {
      Det_Int<TVector_hr> d;
      for (auto &y : x.image) {
        d.image.push_back(TVector_hr(y.begin(), y.end()));
      }
      result.push_back(d);
    }
    impl_->data.image_tot = result;
  }

  template <typename T>
  std::vector<DetInt<typename Output<T>::vector_type>> Output<T>::get_image_coh() const {
    std::vector<DetInt<vector_type>> result;
    for (auto &x : impl_->data.image_coh) {
      DetInt<vector_type> d;
      for (auto &y : x.image) {
        d.image.push_back(vector_type(y.begin(), y.end()));
      }
      result.push_back(d);
    }
    return result;
  }

  template <typename T>
  void Output<T>::set_image_coh(const std::vector<DetInt<vector_type>>& image_coh) {
    typedef typename Output_Multislice<T>::TVector_hr vector_type;
    host_vector<Det_Int<vector_type>> result;
    for (auto &x : image_coh) {
      Det_Int<vector_type> d;
      for (auto &y : x.image) {
        d.image.push_back(vector_type(y.begin(), y.end()));
      }
      result.push_back(d);
    }
    impl_->data.image_coh = result;
  }

  template <typename T>
  std::vector<typename Output<T>::vector_type> Output<T>::get_m2psi_tot() const {
    std::vector<vector_type> result;
    for (auto &x : impl_->data.m2psi_tot) {
      result.push_back(vector_type(x.begin(), x.end()));
    }
    return result;
  }

  template <typename T>
  void Output<T>::set_m2psi_tot(const std::vector<vector_type>& m2psi_tot) {
    typedef typename Output_Multislice<T>::TVector_hr TVector_hr;
    host_vector<TVector_hr> result;
    for (auto &x : m2psi_tot) {
      result.push_back(TVector_hr(x.begin(), x.end()));
    }
    impl_->data.m2psi_tot = result;
  }

  template <typename T>
  std::vector<typename Output<T>::vector_type> Output<T>::get_m2psi_coh() const {
    std::vector<vector_type> result;
    for (auto &x : impl_->data.m2psi_coh) {
      result.push_back(vector_type(x.begin(), x.end()));
    }
    return result;
  }

  template <typename T>
  void Output<T>::set_m2psi_coh(const std::vector<vector_type>& m2psi_coh) {
    typedef typename Output_Multislice<T>::TVector_hr TVector_hr;
    host_vector<TVector_hr> result;
    for (auto &x : m2psi_coh) {
      result.push_back(TVector_hr(x.begin(), x.end()));
    }
    impl_->data.m2psi_coh = result;
  }

  template <typename T>
  std::vector<typename Output<T>::complex_vector_type> Output<T>::get_psi_coh() const {
    std::vector<complex_vector_type> result;
    for (auto &x : impl_->data.psi_coh) {
      result.push_back(complex_vector_type(x.begin(), x.end()));
    }
    return result;
  }

  template <typename T>
  void Output<T>::set_psi_coh(const std::vector<complex_vector_type>& psi_coh) {
    typedef typename Output_Multislice<T>::TVector_hc TVector_hc;
    host_vector<TVector_hc> result;
    for (auto &x : psi_coh) {
      result.push_back(TVector_hc(x.begin(), x.end()));
    }
    impl_->data.psi_coh = result;
  }

  template <typename T>
  std::vector<typename Output<T>::vector_type> Output<T>::get_V() const {
    std::vector<vector_type> result;
    for (auto &x : impl_->data.V) {
      result.push_back(vector_type(x.begin(), x.end()));
    }
    return result;
  }

  template <typename T>
  void Output<T>::set_V(const std::vector<vector_type>& V) {
    typedef typename Output_Multislice<T>::TVector_hr TVector_hr;
    host_vector<TVector_hr> result;
    for (auto &x : V) {
      result.push_back(TVector_hr(x.begin(), x.end()));
    }
    impl_->data.V = result;
  }

  template <typename T>
  std::vector<typename Output<T>::complex_vector_type> Output<T>::get_trans() const {
    std::vector<complex_vector_type> result;
    for (auto &x : impl_->data.trans) {
      result.push_back(complex_vector_type(x.begin(), x.end()));
    }
    return result;
  }

  template <typename T>
  void Output<T>::set_trans(const std::vector<complex_vector_type>& trans) {
    typedef typename Output_Multislice<T>::TVector_hc TVector_hc;
    host_vector<TVector_hc> result;
    for (auto &x : trans) {
      result.push_back(TVector_hc(x.begin(), x.end()));
    }
    impl_->data.trans = result;
  }

  template <typename T>
  std::vector<typename Output<T>::complex_vector_type> Output<T>::get_psi_0() const {
    std::vector<complex_vector_type> result;
    for (auto &x : impl_->data.psi_0) {
      result.push_back(complex_vector_type(x.begin(), x.end()));
    }
    return result;
  }

  template <typename T>
  void Output<T>::set_psi_0(const std::vector<complex_vector_type>& psi_0) {
    typedef typename Output_Multislice<T>::TVector_hc TVector_hc;
    host_vector<TVector_hc> result;
    for (auto &x : psi_0) {
      result.push_back(TVector_hc(x.begin(), x.end()));
    }
    impl_->data.psi_0 = result;
  }

  template <typename T>
  std::vector<bool> Output<T>::get_thk_gpu() const {
    return std::vector<bool>(impl_->data.thk_gpu.begin(), impl_->data.thk_gpu.end());
  }

  template <typename T>
  void Output<T>::set_thk_gpu(const std::vector<bool>& thk_gpu) {
    impl_->data.thk_gpu.assign(thk_gpu.begin(), thk_gpu.end());
  }

  template <typename T>
  void Output<T>::gather() {
    impl_->data.gather();
  }
  
  template <typename T>
  void Output<T>::clean_temporal() {
    impl_->data.clean_temporal();
  }
  
  /****************************************************************************
   * The Multislice interface
   ***************************************************************************/

  /* template <typename T, eDevice dev> */
  /* struct MultisliceData<T, dev>::Data { */
  /*   Multislice<T, dev> data; */
  /* }; */

  /* template <typename T, eDevice dev> */
  /* MultisliceData<T, dev>::MultisliceData() */
  /*     : impl_(std::make_unique<Data>()) {} */

  /* template <typename T, eDevice dev> */
  /* const MultisliceData<T, dev>::Data& MultisliceData<T, dev>::internal() const { */
  /*   return *impl_; */
  /* } */
		
  /* template <typename T, eDevice dev> */
  /* void MultisliceData<T, dev>::set_input_data(Input<T> &input, StreamIface<dev> &stream_i, FFTData<T, dev> &fft2_i) { */
  /*   impl_->data.set_input_data( */
  /*       &input.internal().data, */
  /*       &stream_i.internal().data, */
  /*       &fft2_i.internal().data); */
  /* } */
		
  /* template <typename T, eDevice dev> */
  /* void MultisliceData<T, dev>::operator()(Output<T> &output) { */
  /*   impl_->data(output.internal().data); */
  /* } */

  /****************************************************************************
   * Misc function calls
   ***************************************************************************/

  namespace detail {
  
    template <typename T, eDevice dev>
    mt::Output<T> tem_simulation_internal(Input<T>& input_multislice) {

      // Ensure we have the correct function
      MULTEM_ASSERT(input_multislice.get_system_conf().get_device() == dev);

      // Initialise the stream and plan
      mt::Stream<dev> stream(input_multislice.get_system_conf().get_nstream());
      mt::FFT<T, dev> fft_2d;
      fft_2d.create_plan_2d(
          input_multislice.get_grid_2d().ny, 
          input_multislice.get_grid_2d().nx, 
          input_multislice.get_system_conf().get_nstream());

      // Create the multislice structure
      mt::Multislice<T, dev> simulate;
      simulate.set_input_data(
          &input_multislice.internal().data, 
          &stream, 
          &fft_2d);

      // Initialise the output data
      mt::Output<T> output_multislice;
      output_multislice.set_input_data(input_multislice);
      simulate(output_multislice.internal().data);
      stream.synchronize();

      // Finalise the output data
      output_multislice.gather();
      output_multislice.clean_temporal();
      fft_2d.cleanup();

      // If there is an error then throw
      auto err = hipGetLastError();
      if (err != hipSuccess) {
        std::ostringstream msg;
        msg << "CUDA error: %s\n";
        msg << hipGetErrorString(err);
        throw std::runtime_error(msg.str());
      }

      // Return the output multislice
      return output_multislice;
    }

  }

  template <typename T>
  mt::Output<T> tem_simulation(Input<T>& input_multislice) {
    eDevice dev = input_multislice.get_system_conf().get_device();
    return (dev == mt::e_device 
        ? detail::tem_simulation_internal<T, mt::e_device>(input_multislice)
        : detail::tem_simulation_internal<T, mt::e_host>(input_multislice));
  }
  
  /****************************************************************************
   * Explict instantiation of template classes and functions
   ***************************************************************************/
  template class Lens<float>;
  template class Lens<double>;
  
  template class EELS<float>;
  template class EELS<double>;

  template class AtomData<float>;
  template class AtomData<double>;
  
  template class DetectorData<float>;
  template class DetectorData<double>;

  template class ScanningData<float>;
  template class ScanningData<double>;

  template class Input<float>;
  template class Input<double>;

  template class Output<float>;
  template class Output<double>;

  template mt::Output<float> tem_simulation<float>(Input<float>&);
  template mt::Output<double> tem_simulation<double>(Input<double>&);

}  // namespace mt

namespace std {
  
  /****************************************************************************
   * Explict instantiation of template classes and functions from std
   ***************************************************************************/

  template class complex<float>;
  template class complex<double>;

  template class vector<float>;
  template class vector<double>;
  template class vector<complex<float>>;
  template class vector<complex<double>>;

}
