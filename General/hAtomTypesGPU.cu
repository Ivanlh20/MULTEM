#include "hip/hip_runtime.h"
#include <memory.h>
#include "hConstTypes.h"
#include "hgeneralCPU.h"
#include "hgeneralGPU.h"
#include "hPotentialCPU.h"
#include "hAtomTypesGPU.h"
#include "math.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

void CubicPolyCoefHost(int PotPari, sAtomTypesCPU &AtomTypesCPUi, int nR, double *&R, double *&R2, sciVn &ciVR){
	int IntType = 0, Dim = 2;
	double sigma = 0.0, *VR, *dVR;
	double dR2, V, Vn, dV, dVn, m, n;

	VR = new double[nR];
	dVR = new double[nR];
	cPotentialCPU PotentialCPU;
	PotentialCPU.SetAtomTypes(PotPari, AtomTypesCPUi);
	PotentialCPU.SetSigma(sigma);
	PotentialCPU.Vr(IntType, Dim, nR, R, VR, dVR);
	for(int iR=0; iR<nR; iR++){
		ciVR.c0[iR] = VR[iR]/cPotf;
		ciVR.c1[iR] = 0.5*dVR[iR]/(cPotf*R[iR]);
	}

	for(int iR=0; iR<nR-1; iR++){
		dR2 = 1.0/(R2[iR+1]-R2[iR]);
		V = ciVR.c0[iR]; Vn = ciVR.c0[iR+1];
		dV = ciVR.c1[iR]; dVn = ciVR.c1[iR+1];
		m = (Vn-V)*dR2; n = dV+dVn;
		ciVR.c0[iR] = V-ciVR.c0[nR-1];
		ciVR.c2[iR] = (3.0*m-n-dV)*dR2;
		ciVR.c3[iR] = (n-2.0*m)*dR2*dR2;
	}

	delete [] VR; VR = 0;
	delete [] dVR; dVR = 0;
}

// free memory
void cAtomTypesGPU::freeMemory(){
	hipDeviceSynchronize(); // wait to finish the work in the GPU

	Z = 0;
	m = 0;
	A = 0;
	rn_e = 0;
	rn_c = 0;
	ra_e = 0;
	ra_c = 0;
	Rmin = 0;
	Rmax = 0;
	Rmin2 = 0;
	Rmax2 = 0;

	f_sCoefPar_Free(cfegh);
	f_sCoefPar_Free(cfxgh);
	f_sCoefPar_Free(cPrh);
	f_sCoefPar_Free(cVrh);
	f_sCoefPar_Free(cVRh);

	f_sCoefPar_cudaFree(cfeg);
	f_sCoefPar_cudaFree(cfxg);
	f_sCoefPar_cudaFree(cPr);
	f_sCoefPar_cudaFree(cVr);
	f_sCoefPar_cudaFree(cVR);

	if(ns>0){		
		for (int i=0; i<ns; i++){
			Vo[i].sigma = 0;\
			f_sCoefPar_cudaFree(Vo[i].cVr);
			f_sCoefPar_cudaFree(Vo[i].cVi);
		}
		delete [] Vo; Vo = 0;
		ns = 0;
	}
	
	nR = 0;

	delete [] Rh; Rh = 0;
	delete [] R2h; R2h = 0;
	f_sciVn_Free(ciVRh);

	cudaFreen(R);
	cudaFreen(R2);
	f_sciVn_cudaFree(ciVR);
};

// Constructor
cAtomTypesGPU::cAtomTypesGPU(){
	Z = 0;
	m = 0;
	A = 0;
	rn_e = 0;
	rn_c = 0;
	ra_e = 0;
	ra_c = 0;
	Rmin = 0;
	Rmax = 0;
	Rmin2 = 0;
	Rmax2 = 0;

	f_sCoefPar_Init(cfegh);
	f_sCoefPar_Init(cfxgh);
	f_sCoefPar_Init(cPrh);
	f_sCoefPar_Init(cVrh);
	f_sCoefPar_Init(cVRh);

	f_sCoefPar_cudaInit(cfeg);
	f_sCoefPar_cudaInit(cfxg);
	f_sCoefPar_cudaInit(cPr);
	f_sCoefPar_cudaInit(cVr);
	f_sCoefPar_cudaInit(cVR);

	ns = 0;
	Vo = 0;

	nR = 0;

	Rh = 0;
	R2h = 0;
	f_sciVn_Init(ciVRh);

	R = 0;
	R2 = 0;
	f_sciVn_cudaInit(ciVR);
}

// Destructor
cAtomTypesGPU::~cAtomTypesGPU(){
	freeMemory(); // clean GPU memory
}

// Set Atom type
void cAtomTypesGPU::SetAtomTypes(int PotPari, sAtomTypesCPU &AtomTypesCPUi, int nRi, double dRmini){
	freeMemory(); // clean GPU memory

	Z = AtomTypesCPUi.Z;
	m = AtomTypesCPUi.m;
	A = AtomTypesCPUi.A;
	rn_e = AtomTypesCPUi.rn_e;
	rn_c = AtomTypesCPUi.rn_c;
	ra_e = AtomTypesCPUi.ra_e;
	ra_c = AtomTypesCPUi.ra_c;
	Rmin = MAX(rn_c, dRmini);
	Rmax = AtomTypesCPUi.Rmax;
	Rmin2 = Rmin*Rmin;
	Rmax2 = Rmax*Rmax;

	/***************************************************************/
	f_sCoefPar_Malloc(6, cfegh);
	memcpy(cfegh.cl, AtomTypesCPUi.cfeg.cl, 6*cSizeofRD);
	memcpy(cfegh.cnl, AtomTypesCPUi.cfeg.cnl, 6*cSizeofRD);

	f_sCoefPar_Malloc(6, cfxgh);
	memcpy(cfxgh.cl, AtomTypesCPUi.cfxg.cl, 6*cSizeofRD);
	memcpy(cfxgh.cnl, AtomTypesCPUi.cfxg.cnl, 6*cSizeofRD);

	f_sCoefPar_Malloc(6, cPrh);
	memcpy(cPrh.cl, AtomTypesCPUi.cPr.cl, 6*cSizeofRD);
	memcpy(cPrh.cnl, AtomTypesCPUi.cPr.cnl, 6*cSizeofRD);

	f_sCoefPar_Malloc(6, cVrh);
	for (int i=0; i<6; i++)
		cVrh.cl[i] = AtomTypesCPUi.cVr.cl[i]/cPotf;
	memcpy(cVrh.cnl, AtomTypesCPUi.cVr.cnl, 6*cSizeofRD);

	f_sCoefPar_Malloc(6, cVRh);
	for (int i=0; i<6; i++)
		cVRh.cl[i] = AtomTypesCPUi.cVR.cl[i]/cPotf;
	memcpy(cVRh.cnl, AtomTypesCPUi.cVR.cnl, 6*cSizeofRD);

	/***************************************************************/
	f_sCoefPar_cudaMalloc(6, cfeg);
	hipMemcpy(cfeg.cl, cfegh.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cfeg.cnl, cfegh.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	f_sCoefPar_cudaMalloc(6, cfxg);
	hipMemcpy(cfxg.cl, cfxgh.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cfxg.cnl, cfxgh.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	f_sCoefPar_cudaMalloc(6, cPr);
	hipMemcpy(cPr.cl, cPrh.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cPr.cnl, cPrh.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	f_sCoefPar_cudaMalloc(6, cVr);
	hipMemcpy(cVr.cl, cVrh.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cVr.cnl, cVrh.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	f_sCoefPar_cudaMalloc(6, cVR);
	hipMemcpy(cVR.cl, cVRh.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cVR.cnl, cVRh.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	/***********************************************************************************/
	ns = AtomTypesCPUi.ns;
	if(ns>0){
		Vo = new sVoGPU[ns];
		for (int i=0; i<ns; i++){
			Vo[i].sigma = AtomTypesCPUi.Vo[i].sigma;
			f_sCoefPar_cudaMalloc(6, Vo[i].cVr);
			hipMemcpy(Vo[i].cVr.cl, AtomTypesCPUi.Vo[i].cVr.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
			hipMemcpy(Vo[i].cVr.cnl, AtomTypesCPUi.Vo[i].cVr.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

			f_sCoefPar_cudaMalloc(6, Vo[i].cVi);
			hipMemcpy(Vo[i].cVi.cl, AtomTypesCPUi.Vo[i].cVi.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
			hipMemcpy(Vo[i].cVi.cnl, AtomTypesCPUi.Vo[i].cVi.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);
		}
	}

	nR = nRi;

	double dlnr = log(Rmax/Rmin)/double(nR-1);
	Rh = new double [nR];
	R2h = new double [nR];
	for (int i=0; i<nR; i++){
		Rh[i] = Rmin*exp(double(i)*dlnr);
		R2h[i] = Rh[i]*Rh[i];
	}
	f_sciVn_Malloc(nR, ciVRh);
	CubicPolyCoefHost(PotPari, AtomTypesCPUi, nR, Rh, R2h, ciVRh);

	hipMalloc((void**)&R, nR*cSizeofRD);
	hipMemcpy(R, Rh, nR*cSizeofRD, hipMemcpyHostToDevice);
	hipMalloc((void**)&R2, nR*cSizeofRD);
	hipMemcpy(R2, R2h, nR*cSizeofRD, hipMemcpyHostToDevice);

	f_sciVn_cudaMalloc(stnR, ciVR);
	hipMemcpy(ciVR.c0, ciVRh.c0, nR*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(ciVR.c1, ciVRh.c1, nR*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(ciVR.c2, ciVRh.c2, nR*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(ciVR.c3, ciVRh.c3, nR*cSizeofRD, hipMemcpyHostToDevice);
}